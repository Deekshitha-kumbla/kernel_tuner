#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

extern "C"
{
__global__ void vector_add(const {{ real_type }} * a,const {{ real_type }} * b, {{ real_type }} * c, int size)
{
    int i = (blockIdx.x * block_size_x) + threadIdx.x;

    {% for tile in range(tiling_x) %}
    {% set offset = block_size_x * tile %}
    if ( i + {{ offset }} < (size / {{ vector_size }}) )
    {
        c[i + {{ offset }}] = a[i + {{ offset }}] + b[i + {{ offset }}];
    }
    {% endfor %}
}
}